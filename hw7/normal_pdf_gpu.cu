
#include <hip/hip_runtime.h>
extern "C"   // ensure function name to be left alone
{

    __global__ void normal_pdf_gpu(const double *x, double *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
        if(i<n)  y[i] = exp(-0.5*x[i]*x[i])*rsqrt(2.0*M_PI);
    }

    __global__ void sum_gpu(double *y, double *sumptr, unsigned int n,
            unsigned int n_subsums, unsigned int percore)
    {
        // assumes a 2-d grid of 1-d blocks
        unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x
            + threadIdx.x;
        unsigned int j = i * percore; // first element that this thread will take care of
        unsigned int k;

        if (i >= n_subsums)
            return;

        sumptr[i] = 0.0;
        for (k = 0; k < percore; k++) {
            if (j + k < n)
                sumptr[i] += y[j + k];
        }
    }

    __global__ void get_sum_gpu(double *y, double *sumptr)
        // copy the result into a smaller array
    {
        sumptr[0] = y[0];
    }
}


/* vim: set sw=4 sts=4 et : */
