
#include <hip/hip_runtime.h>
extern "C"   // ensure function name to be left alone
{
    __global__ void sum_gpu(double *y, double *sumptr, unsigned int n, unsigned int n_subsums, unsigned int percore)
    {
        // assumes a 2-d grid of 1-d blocks
        unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
        unsigned int j = i * percore;  // first element that this thread will take care of
        unsigned int k;

        if (i >= n_subsums)
            return;

        sumptr[i] = 0.0;
        for (k = 0; k < percore; k++) {
            if (j + k < n)
                sumptr[i] += y[j + k];
        }
    }
}


/* vim: set sw=4 sts=4 et : */
